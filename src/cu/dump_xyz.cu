#include "ff/image.h"
#include "ff/molecule.h"
#include "ff/atom.h"

namespace tinker {
    void fromGPU_xyz(real* h_x, real* h_y, real* h_z)
    {
        hipMemcpy(h_x, x, n * sizeof(real), hipMemcpyDeviceToHost);
        hipMemcpy(h_y, y, n * sizeof(real), hipMemcpyDeviceToHost);
        hipMemcpy(h_z, z, n * sizeof(real), hipMemcpyDeviceToHost);
    }

    void toGPU_xyz(real* h_x, real* h_y, real* h_z)
    {
        hipMemcpy(x, h_x, n * sizeof(real), hipMemcpyHostToDevice);
        hipMemcpy(y, h_y, n * sizeof(real), hipMemcpyHostToDevice);
        hipMemcpy(z, h_z, n * sizeof(real), hipMemcpyHostToDevice);
    }
}
